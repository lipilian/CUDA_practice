#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "common.h"
#include "cuda_common.cuh"

__global__ void reduction_neighbored_paris_improved(int* input, int* temp, int size) {
	int tid = threadIdx.x;
	int gid = blockDim.x * blockIdx.x + tid;

	//local data block pointer
	//int* i_data = input + blockDim.x * blockIdx.x;

	if (gid > size)
		return;

	for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
		//int index = 2 * offset * tid;

		//if (index < blockDim.x) {
		//	i_data[index] += i_data[index + offset];
		//}
		if (tid < offset) {
			input[gid] += input[gid + offset];
		}
		__syncthreads();
	}
	if (tid == 0) {
		temp[blockIdx.x] = input[gid];
	}
}

int main(int argc, char** argv)
{	
	printf("Running neighbored paris reduction kernel \n");

	int size = 1 << 27; // 128Mb of data
	int byte_size = size * sizeof(int);
	int block_size = 128;

	int* h_input, * h_ref;
	h_input = (int*)malloc(byte_size);
	initialize(h_input, size, INIT_RANDOM);

	//GET CPU RESULT
	int cpu_result = reduction_cpu(h_input, size);

	dim3 block(block_size);
	dim3 grid(size / block.x);

	printf("Kernel lauch parameters | grid :%d, block: %d", grid.x, block.x);

	int temp_array_byte_size = sizeof(int) * grid.x;

	h_ref = (int*)malloc(temp_array_byte_size);

	int* d_input, * d_temp;

	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

	hipMemset(d_temp, 0, temp_array_byte_size);
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	reduction_neighbored_paris_improved << <grid, block >> > (d_input, d_temp, size);

	gpuErrchk(hipDeviceSynchronize());

	hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost);

	int gpu_result = 0;

	for (int i = 0; i < grid.x; i++) {
		gpu_result += h_ref[i];
	}

	compare_results(gpu_result, cpu_result);

	hipFree(d_temp);
	hipFree(d_input);

	free(h_ref);
	free(h_input);
	
	gpuErrchk(hipDeviceReset());
	return 0;
}